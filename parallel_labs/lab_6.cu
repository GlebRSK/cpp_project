#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <ctime>
#include <math.h>
//#include <hip/hip_runtime.h>

using namespace std;

__global__ void Matrix_A (double *dA, double *dx, int N) {
        int j = blockIdx.x*blockDim.x + threadIdx.x;
        int i = blockIdx.x*blockDim.y + threadIdx.y;
        
        int Nd = (int)(0.15*N);
        
        if (i <= j + Nd && i >= j - Nd) {
            dA[i + j*N] = pow(sin(dx[j])*cos(dx[i]), 2) + (double)N*D(i,j); 
        } else {
            dA[i + j*N]    = 0;        
        }
}

__global__ void AX (double *dAX, double *dA, double *dX, int N) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    double sum = 0;
    
    for (int j = 0; j < N; j++) {
        sum += dA[i + j*N]*dX[j];
    }
    
    dAX[i] = sum;
}

__global__ void PHI (double *dPhi, double *dAX, double *dF) {
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        dPhi[i] = dAX[i] - dF[i];
}

__global__ void D_PHI (double *dL, double *dX0, int N) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    double sum1 = sum2 = 0.; int k,k1,k2,Nd = (int)(0.15*N);
    if ( i <= j + Nd && i >= j - Nd ) {
        
        if ( i >= 0 && i <= Nd ) {
            k1 = 0; k2 = i+Nd + 1;
        } // область 1
        
        if ( i >= Nd+1 && i < N-Nd ) {
            k1 = i-Nd; k2 = i+Nd+1;
        } // область 2
        
        if ( i >= N-Nd && i < N ) {
            k1 = i-Nd; k2 = N;
        } // область 3
        
        for ( k = k1; k < k2; k++ ){
            sum1 += D(k,j)*(pow(sin(dX0[i])*cos(dX0[k]),2.)+D(i,k)*(double)N);
            sum2 += dX0[k]*(sin(2.*dX0[i])*pow(cos(dX0[k]),2.)*D(i,j)-
            sin(2.*dX0[k])*pow(sin(dX0[i]),2.)*D(k,j));
        } // k
        dL[i+j*N] = sum1 + sum2; // dLT !
    } else {
        dL[i+j*N] = 0.;
    }
}

__global__ void Solve_G (double *dX0, double *dX1, double *dV0, double tau) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    dX1[i] = dX0[i] + tau * dV0[i];
}

__global__ void Eps_G (double *dX0, double *dX1, double *d_dX ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_dX[i] = abs (dX0[i] - dX1[i]);
    dX0[i] = dX1[i];
}
    
int main() {

    while ( eps_G > EPS_G ) {
        Matrix_A <<< nBlk_MtrxA, nTid_MtrxA >>> ( dA, dX0, N );
        AX <<< nBlk_L, nTid_L >>> ( dAX, dA, dX0, N );
        PHI <<< nBlk_L, nTid_L >>> ( dPhi, dAX, dF );
        D_PHI <<< nBlk_MtrxA, nTid_MtrxA >>> ( dL, dX0, N );
        hipMemset ( dV0, 1, mem_sizeX ); 
        eps_L = 1.;
        
        while ( eps_L > EPS_L ) {
            Solve_L <<< nBlk_L, nTid_L >>> ( dL, dPhi, dV0, dV1, N );
            Eps_L <<< nBlk_L, nTid_L >>> ( dV0, dV1, d_dV, N );
            hipMemcpy ( h_dV, d_dV, mem_sizeX, hipMemcpyDeviceToHost );
            eps_L=0.; 
            for ( j = 0; j < N; j++ )
                eps_L += h_dV[j]; 
            eps_L = eps_L / N;
        } //while_L

        Solve_G <<< nBlk_L, nTid_L >>> ( dX0, dX1, dV0, tau );
        Eps_G <<< nBlk_L, nTid_L >>> ( dX0, dX1, d_dX );
        hipMemcpy ( h_dX, d_dX, mem_sizeX, hipMemcpyDeviceToHost );
        eps_G=0.; 
        
        for ( k = 0; k < N; k++ )
            eps_G += h_dX[k]; 
        eps_G = eps_G / N;
    } 
    return 0;
}