
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

#define blockSize 32

__global__ void Multiplication (int *dA, int *dB, int *dC, int aSize, int bSize, int cSize) {
    int t = blockIdx.y*blockSize + threadIdx.y;
    int v = blockIdx.x*blockSize + threadIdx.x;
    
    int summ = 0;
    
    for (int i = 0; i < (blockSize + aSize -1)/ blockSize; i++) {
        for (int j = 0; j < blockSize; j++) {
            if ((i*blockSize + j < aSize && t < aSize) && (i*blockSize + j < bSize && v < bSize)) {
                summ += dA[t*aSize + i*blockSize + j]*dB[(i*blockSize + j)*bSize + v];
            }
        }
    }
    if (t < cSize && v < cSize)
        dC[t*cSize + v] = summ;
        
}

void fillingMatrix (int **a, int dim1, int dim2, bool b) {
    if (b) {
        for (int i = 0; i < dim1; i++) {
            for (int j = 0; j < dim2; j++) {
                a[i][j] = rand() % 15 - 5;
            }
        }
    } 
    else {
        for (int i = 0; i < dim1; i++) {
            for (int j = 0; j < dim2; j++) {
                a[i][j] = 0;
            }
        }
    }
}

void twiceDimToOneDim(int **dim2, int *dim1, int x, int y, bool needTranspose) {
    if (!needTranspose) {
        for (int i = 0; i < y; i++) {
            for (int j = 0; j < x; j++)    dim1[i*x + j] = dim2[i][j];
        }
    }
    else {
        for (int i = 0; i < y; i++) {
            for (int j = 0; j < x; j++)    dim1[i*x + j] = dim2[j][i];
        }
    }
}

void multiply (int *a, int *b, int *c, int dimA, int dimB) {
    int summ = 0;
    for (int i = 0; i < dimA; i++) {
        for (int j = 0; j < dimB; j++) {
            for (int e = 0; e < blockSize*blockSize; e++) {
                summ += a[j*blockSize*blockSize + e]*b[j*blockSize*blockSize + e];
            }
        }
    }
}

void printMatrix (int **a, int b, int c, string matrixName) {
    
    for (int i = 0; i < b; i++) {
        for (int j = 0; j < c; j++) {
            cout << a[i][j] << " ";
        } cout << endl;
    }
    cout << endl;
}

int main() {
    srand(time(0));

    int dimyA = 1024, dimxA = 1024, dimxB = 1024, dimyB = 1024;
    int deepC= dimxA/blockSize;
    int **a = new int*[dimyA];
    int **b = new int*[dimyB];
    int ***c = new int**[dimyA];
    int **ct = new int*[dimyA];
    int *devA, *devB, *devC;
    int *oneDimA = new int[dimyA*dimxA];
    int *oneDimB = new int[dimxB*dimyB];
    int *oneDimC = new int[dimyA*dimxB];

    for (int i = 0; i < dimyA; i++) {
        a[i] = new int[dimxA];
        c[i] = new int*[dimxB];
        ct[i] = new int[dimxB];
        for (int j = 0; j < dimxB; j++) {
            c[i][j] = new int[deepC];
        }
    }
    
    for (int i = 0; i <dimyB; i++) {
        b[i] = new int[dimxB];
    }
    
    
    clock_t startCPU = clock();
    
    fillingMatrix(a, dimyA, dimxA, true);
    fillingMatrix(b, dimyB, dimxB, true);
    fillingMatrix(ct, dimyA, dimxB, false);
    
    clock_t endCPU = clock();
    cout << (float)(endCPU - startCPU)/CLOCKS_PER_SEC << endl;
    
    startCPU = clock();
    for (int i = 0; i < dimxA; i++) {
        for (int j = 0; j < dimxA; j++) {
            for (int e = 0; e < dimxA; e++) {
                ct[i][j] += a[i][e]*b[e][j];    
            }
        }
    }
    
    endCPU = clock();
    cout << (float)(endCPU - startCPU)/CLOCKS_PER_SEC << endl;

    startCPU = clock();
    int countLayer = 0;
    while (countLayer < deepC) {
        for (int i = 0; i < dimyA; i += blockSize) {
            for (int j = 0; j < dimxB; j += blockSize) {
                for (int e = 0; e < blockSize; e++) {
                    for (int o = 0; o < blockSize; o++) {
                        for (int u = 0; u < blockSize; u++) {
                            c[i + e][j + o][0] += a[i + e][j + u]*b[i + u][j + o];
                        }
                    }
                }
            }
        }
        countLayer++;
    }

    endCPU = clock();
    cout << (float)(endCPU - startCPU)/CLOCKS_PER_SEC << endl;
    //printMatrix(ct, dimyA, dimxB, "C");
    
    twiceDimToOneDim(a, oneDimA, dimyA, dimxA, false);
    twiceDimToOneDim(b, oneDimB, dimyB, dimxB, false);
    twiceDimToOneDim(ct, oneDimC, dimyB, dimxB, false);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    hipMalloc ((void**) &devA, sizeof (int)*dimyA*dimxA);
    hipMalloc ((void**) &devB, sizeof (int)*dimxB*dimyB);
    hipMalloc ((void**) &devC, sizeof (int)*dimyA*dimxB);
    
    hipMemcpy(devA, oneDimA, sizeof (int)*dimyA*dimxA, hipMemcpyHostToDevice);
    hipMemcpy(devB, oneDimB, sizeof (int)*dimyB*dimxB, hipMemcpyHostToDevice);

    dim3 dimBlock(dimyA, dimyA);
    dim3 dimGrid(blockSize, blockSize);
    
    Multiplication <<<dimGrid, dimBlock>>> (devA, devB, devC, dimyA, dimyB, dimxA);
    
    hipMemcpy(oneDimC, devC, sizeof (int)*dimyA*dimxB, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float eps;
    
    hipEventElapsedTime(&eps, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    
    /*for (int i = 0; i < dimyA; i++) {
        for (int j = 0; j < dimxA; j++) {
            cout << oneDimC[i*dimxA + j] << " ";
        } cout << endl;
    }*/
 

    cout << eps/1000 << endl;
    
    return 0;
}
