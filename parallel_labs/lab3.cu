
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <ctime>
#include <math.h>
//#include <cuda_runtime.h>

using namespace std;
__global__ void Solve (double *a, double *c, double *x0, double *x1, int N) {

    double aa = 1, sum = 0;
    int t = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < N; i++) {
        //sum += a[i + t*N] * x0[i];
        sum += a[t + i*N] * x0[i];
        if (i == t)
            aa = a[i + t*N];
    }
    x1[t] = x0[t] + (c[t] - sum) / aa;
}

__global__ void Eps (double *x0, double *x1, double *delta, int N) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    delta[i] = abs (x0[i] - x1[i]);
    x0[i] = x1[i];
}

void fillingOneDIM(double *a, int size, bool b) {

    if (b) {

        for (int i = 0; i < size; i++) {
            a[i]  = (double)(-10 + rand()*10)/RAND_MAX;
        }
    }
    else {

        for (int i = 0; i < size; i++) {
            a[i] = 1;
        }
    }
}

int main() {
    //srand(time(0));
    double EPS = 1e-8, eps = 1;
    int N = 2048, size = N*N, N_Thread = 512, numIteration = 0;
    int mem_sizeA = sizeof (double)*size;
    int mem_sizeX = sizeof (double)*N;

    double *devA, *devC, *devX0, *devX1, *devDelta;
    double *hA = (double*) malloc (mem_sizeA);
    double *hC = (double*) malloc (mem_sizeX);
    double *hX = (double*) malloc (mem_sizeX);
    double *hX0 = (double*) malloc (mem_sizeX);
    double *hX1 = (double*) malloc (mem_sizeX);
    double *hDelta = (double*) malloc (mem_sizeX);

    fillingOneDIM (hA, size, true);
    fillingOneDIM (hC, N, true);
    fillingOneDIM (hX, N, false);
    fillingOneDIM (hX0, N, false);

    for (int i = 0; i < size; i += (N + 1)) {

        if (hA[i] == 0) {
            hA[i] = 1;
        }
        hA[i] = fabs(hA[i])*70100;
    }

    hipMalloc ((void**) &devA, mem_sizeA);
    hipMalloc ((void**) &devC, mem_sizeX);
    hipMalloc ((void**) &devX0, mem_sizeX);
    hipMalloc ((void**) &devX1, mem_sizeX);
    hipMalloc ((void**) &devDelta, mem_sizeX);

    

    hipMemcpy (devA, hA, mem_sizeA, hipMemcpyHostToDevice);
    hipMemcpy (devC, hC, mem_sizeX, hipMemcpyHostToDevice);
    hipMemcpy (devX0, hX0, mem_sizeX, hipMemcpyHostToDevice);

    int N_Blocks = 4;
    
    float timerValueGPU;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord (start, 0);
    
    do {
        numIteration++;
        Solve <<<N_Blocks, N_Thread>>> (devA, devC, devX0, devX1, N);
        Eps <<<N_Blocks, N_Thread>>> (devX0, devX1, devDelta, N);
        hipMemcpy (hDelta, devDelta, mem_sizeX, hipMemcpyDeviceToHost);

        eps = 0;
        for (int i = 0; i < N; i++) {
            eps += hDelta[i];
        }
        eps /= N;
    } while(eps > EPS);
//    cout << numIteration << endl;
    hipMemcpy (hX1, devX0, mem_sizeX, hipMemcpyDeviceToHost);

    hipEventRecord (stop, 0);
    hipEventSynchronize (start);
    hipEventSynchronize (stop);
        hipEventElapsedTime (&timerValueGPU, start, stop);
    cout << "time GPU: " << timerValueGPU/1000000 << endl;

    hipFree(devX0);
    hipFree(devC);
    hipFree(devX0);

    double sum = 0;
    numIteration = 0;

    float timeStart = clock()/(float)CLOCKS_PER_SEC;
    eps = 1;

    while (eps > EPS) {
        numIteration++;
        cout << numIteration << endl;
        for (int i = 0; i < N; i++) {
            sum = 0;
            double aa = 1;

            for (int j = 0; j < N; j++) {
                sum += hA[i + j*N]*hX0[j];
                if (i == j)
                    aa = hA[j + i*N];
            }
            hX1[i] = hX0[i] + (hC[i] - sum) / aa;
        }
        eps = 0;

        for (int i = 0; i < N; i++) {
            hDelta[i] = fabs (hX0[i] - hX1[i]);
            eps += hDelta[i];
            hX0[i] = hX1[i];
        }

        eps /= N;
    }

    float timeStop = clock()/(float)CLOCKS_PER_SEC;
    cout << "time CPU: " << (float)timeStop - timeStart << endl;
    return 0;
}
